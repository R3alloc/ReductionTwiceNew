#include "kernel.cuh"


void cudaResultCheck(hipError_t result,char* fileName, char* functionName, int lineNum)
{
	if (result != hipSuccess)
	{
		hipError_t error = hipGetLastError();
		printf("*CUDA error in file %s, \n*function %s, \n*line %d: %s\n",fileName, functionName, lineNum, hipGetErrorString(error));
	}
	return;
}

void cudaInit()
{
	//hipError_t result = hipSetDevice(1);
	hipError_t result = hipSetDevice(0);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

void devMalloc(int** devData, int dataSize)
{
	hipError_t result = hipMalloc((void**)devData, dataSize);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

