#include "kernel.cuh"


void cudaResultCheck(hipError_t result,char* fileName, char* functionName, int lineNum)
{
	if (result != hipSuccess)
	{
		hipError_t error = hipGetLastError();
		printf("*CUDA error in file %s, \n*function %s, \n*line %d: %s\n",fileName, functionName, lineNum, hipGetErrorString(error));
	}
	return;
}

void cudaInit()
{
	//hipError_t result = hipSetDevice(1);
	hipError_t result = hipSetDevice(0);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

void devMalloc(int** devData, int dataSize)
{
	hipError_t result = hipMalloc((void**)devData, dataSize);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

void hostRegister(int* imgData, int dataSizeByte)
{
	hipError_t result = hipHostRegister(imgData,dataSizeByte,hipHostRegisterDefault);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

void hostFree(int* imgData)
{
	hipError_t result =  hipHostUnregister(imgData);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}
