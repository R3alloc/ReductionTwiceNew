#include "hip/hip_runtime.h"
#include "kernel.cuh"
//using namespace std;


void cudaResultCheck(hipError_t result,char* fileName, char* functionName, int lineNum)
{
	if (result != hipSuccess)
	{
		hipError_t error = hipGetLastError();
		printf("*CUDA error in file %s, \n*function %s, \n*line %d: %s\n",fileName, functionName, lineNum, hipGetErrorString(error));
	}
	return;
}

void cudaInit(vector<int>& iGPU,
	vector<void*>& stream)
{
	hipDeviceReset();
	//hipError_t result = hipSetDevice(1);
	int deviceNum;
	hipGetDeviceCount(&deviceNum);
	for (int i = 0; i < deviceNum; i++)
	{
		iGPU.push_back(i);
		for (int j = 0; j < NUM_STREAM_PER_DEVICE; j++)
		{
			hipStream_t* newStream = new hipStream_t;
			stream.push_back((void*)newStream);
		}
	}
	//hipError_t result = hipSetDevice(0);
	//cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

void cudaEndUp(vector<int>& iGPU,
	vector<void*>& stream)
{
	
	for (int j = 0; j < stream.size(); j++)
	{
		delete stream[j];
	}	

	/*����hipDeviceReset��ʹ�û���Ҫ���о�һ�� ��current device��current process���й�
	for (int i = 0; i < iGPU.size(); i++)
	{
		hipDeviceReset();
	}
	*/
}

void devMalloc(int** devData, int dataSize)
{
	hipError_t result = hipMalloc((void**)devData, dataSize);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

void hostRegister(int* imgData, int dataSizeByte)
{
	//ע��ʹ��hipMemcpyAsync��ʱ���õ���stream��Ҫ��ʹ����ҳ�ڴ档
	//��������ַ���Ӧ�ö����ԣ�һ���ǽ��Ѿ������һ���ڴ�registerΪ��ҳ�ڴ�
	//��һ����ֱ�ӷ����µ��ڴ�Ϊ��ҳ�ڴ�
	hipError_t result = hipHostRegister(imgData,dataSizeByte,hipHostRegisterDefault);
	//hipError_t result = hipHostAlloc((void**)&imgData, dataSizeByte, hipHostMallocDefault);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}

void hostFree(int* imgData)
{
	hipError_t result =  hipHostUnregister(imgData);
	cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
	return;
}


void substract(
	vector<void*>& stream,	//����ʹ��vector��Ҫ��ͷ�ļ��а�������
	vector<int>& iGPU,
	int* imgData,	//imgData���д洢��IMAGE_TOTAL_NUM����Ƭ�����ǻ��ڶ�GPU����stream���У��ٴβ��һ��
	int idim,		//reMask�����ֵ�������_para.size,��ͼ��һ���ߵĳ��ȡ�
	int nImg,		//����substract���������ͼƬ��������function.cpp����Ҳ��һ��batch��һ���СΪIMAGE_BATCH
	int nGPU
)
{
	//LOG(INFO) << "Subtract begin.";

	//ÿ��GPU��Ӧһ��int*
	//��дdelete�ռ����
	//��һ���ǲ���Ҫ�� ��reMask��������ôһ����Ϊ�˸�ÿ̨GPUһ��mask�Է������
	//int** devSubstract = new int*[nGPU];

	//һ��image�����ص�����
	size_t imgSizeRL = idim * idim;

	//����stream������
	int nStream = nGPU * NUM_STREAM_PER_DEVICE;

	//������鵱�д洢����ָ�룬ÿ��ָ�붼ָ��һ���οռ��ַ���ܴ洢BATCH_SIZE��image��
	//int** dev_image_buf = new int*[nStream];
	//int* dev_image_buf[nStream];
	int** dev_image_buf = (int**)malloc(sizeof(int*)*nStream);

	int threadInBlock = (idim > THREAD_PER_BLOCK) ? THREAD_PER_BLOCK : idim;

	//base Stream
	int baseS;

	int nImgBatch = 0;
	int smidx = 0;

	//Ϊÿ̨GPU�ϵ�ÿ��������ռ䣬����BATCH_SIZE����Ƭ��ͼƬ
	for (int n = 0; n < nGPU; n++)
	{
		baseS = n * NUM_STREAM_PER_DEVICE;
		hipSetDevice(iGPU[n]);

		for (int i = 0; i < NUM_STREAM_PER_DEVICE; i++)
		{
			cout << "Allocate memory for GPU[" << n << "],stream[" << i << "]" << endl;
			hipError_t result = hipMalloc((void**)&dev_image_buf[i + baseS], BATCH_SIZE * imgSizeRL * sizeof(int));
			cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
		}
	}

	//LOG(INFO) << "alloc memory done, begin to calculate...";

	//���ͷ�
	int* partial;
	hipMalloc((void**)&partial, THREAD_PER_BLOCK * sizeof(int));

	for (int i = 0; i < nImg;)
	{

		//��GPU���� ������̨����ֻ��һ̨GPU
		for (int n = 0; n < nGPU; n++)
		{
			//ע�����������ѭ����Ҫ©��
			if (i > nImg)
			{
				break;
			}

			//�趨device���ҿ��ٿռ䣬������
			hipSetDevice(iGPU[n]);
			//hipError_t result = hipMalloc((void**)&devSubstract[n], imgSizeRL * sizeof(int));
			//cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);

			//���ڵ�ǰGPU��base stream
			baseS = n * NUM_STREAM_PER_DEVICE;
			nImgBatch = (i + BATCH_SIZE < nImg) ? BATCH_SIZE : (nImg - i);
			
			//����Ĳ���˵��imgData���е������ǿ���������ȡ��
			for (int idx = 0; idx < imgSizeRL; idx++)
			{
				cout << imgData[(nImgBatch-1) * imgSizeRL + idx] << " ";
			}
			//dev_image_buf[nImgBatch * imgSizeRL - 1] = 0;
			//dev_image_buf[nImgBatch * imgSizeRL ] = 0;
			hipError_t result = hipMemcpy(
				dev_image_buf[smidx + baseS],
				imgData + i * imgSizeRL,	//ע��ָ���ƫ����������ȥ��sizeof(int)
				//&(imgData[i * imgSizeRL]),
				nImgBatch * imgSizeRL * sizeof(int),
				hipMemcpyHostToDevice);
			cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
			
			result = hipMemcpyAsync(
				dev_image_buf[smidx + baseS],
				imgData + i * imgSizeRL,	//ע��ָ���ƫ����������ȥ��sizeof(int)
				//&(imgData[i * imgSizeRL]),
				nImgBatch * imgSizeRL * sizeof(int),
				hipMemcpyHostToDevice);//����stream���еĴ洢����Ϊvoid*��������Ҫ��ת��ָ�������ٽ����á�
			
			cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
			hipError_t error = hipGetLastError();
			printf("*CUDA error �� %s\n", hipGetErrorString(error));
			//�����ݴ�host������device��
			//�첽����
			//hipError_t result = hipMemcpyAsync(
			result = hipMemcpyAsync(
				dev_image_buf[smidx + baseS],
				imgData + i * imgSizeRL,	//ע��ָ���ƫ����������ȥ��sizeof(int)
				//&(imgData[i * imgSizeRL]),
				nImgBatch * imgSizeRL * sizeof(int),
				hipMemcpyHostToDevice,
				*((hipStream_t*)(stream[smidx + baseS]))	//����stream���еĴ洢����Ϊvoid*��������Ҫ��ת��ָ�������ٽ����á�
			);
			cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);

			for (int r = 0; r < nImgBatch; n++)
			{
				//����ƫ����
				long long shiftRL = (long long)r * imgSizeRL;
				//һ��ֻ����һ����Ƭ,��������֮������ֱ��д��dev_image_buf����
				/*
				kernel_substract <<<
					idim,			//��Ϊidim��block
					threadInBlock,	//һ��block������threadInBlock���߳�
					0,
					*((hipStream_t*)stream[smidx + baseS])>>>(
						dev_image_buf[smidx + baseS],	//���ﱣ���˴�imgData���濽�����������ݣ������޸�֮��ǵ�д��ȥ
						r,								//�������batch���еĵ�r����Ƭ
						idim,							//һ����Ƭ�ĳ���/���
						imgSizeRL						//һ����Ƭ��ʵ�ռ䵱����ռ�ݵ����ص���
						);
				*/
				//�����ֵ
				int mean;
				int stddev;
				
				Reduction_mean(&mean, 
					partial, 
					dev_image_buf[smidx + baseS],
					imgSizeRL,
					idim, 
					THREAD_PER_BLOCK, 
					*((hipStream_t*)stream[smidx + baseS]));
				//�����׼��
				//TODO

				//����dev_image_buf���е�����
				//TODO

				cout <<"Image "<<r<<": mean = " << mean << endl;


			}



			//һ��batch�����е���Ƭ�������֮��Ӧ��д��imgData����
			//TODO

			i += nImgBatch;

		}

		smidx = (smidx + 1) % NUM_STREAM_PER_DEVICE;
	}

	
	hipFree(partial);
	//delete[] devSubstract;
	
	//Ϊÿ̨GPU�ϵ�ÿ��������ռ䣬����BATCH_SIZE����Ƭ��ͼƬ
	for (int n = 0; n < nGPU; n++)
	{
		baseS = n * NUM_STREAM_PER_DEVICE;
		hipSetDevice(iGPU[n]);

		for (int i = 0; i < NUM_STREAM_PER_DEVICE; i++)
		{
			cout << "Free memory for GPU[" << n << "],stream[" << i << "]" << endl;
			hipError_t result = hipFree(dev_image_buf[i + baseS]);
			cudaResultCheck(result, __FILE__, __FUNCTION__, __LINE__);
		}
	}
}

/*һ��ֻ����һ����Ƭ,��������֮������ֱ��д��dev_image_buf����
	kernel_substract << <
		idim,			//��Ϊidim��block
		threadInBlock,	//һ��block������threadInBlock���߳�
		0,
		*((hipStream_t*)stream[smidx + baseS]) >> > (
			dev_image_buf[smidx + baseS],	//���ﱣ���˴�imgData���濽�����������ݣ������޸�֮��ǵ�д��ȥ
			r,								//�������batch���еĵ�r����Ƭ
			idim,							//һ����Ƭ�ĳ���/���
			imgSizeRL						//һ����Ƭ��ʵ�ռ䵱����ռ�ݵ����ص���
			);
*/
/*
__global__ void kernel_substract(
	int* dev_image,
	int imgIdx,
	int dim,
	size_t imgSizeRL
	)
{
	//grid�е�block��һά��֯��block�е��߳�Ҳ��һά��֯
	int tid = threadIdx.x + blockDim.x * blockIdx.x;



}
*/

__global__ void
Reduction1_kernel(int* out, const int* in, size_t N)
{
	//�������Ĵ�С��blockSize�йأ�Ҳ����blockDim.x��
	//ע��������������ﶨ���ʱ����Ȼû��ָ����С�������ڵ������kernel��ʱ����һ���˺�������������������kernel�ڲ�ʹ�ù����ڴ�Ĵ�С��
	extern __shared__ int sPartials[];
	int sum = 0;
	//tid�ǵ�ǰ�߳��ڵ�ǰblock�е�����
	const int tid = threadIdx.x;
	//i�ǵ�ǰ�߳��������߳��е�����
	//i�Ĳ�����grid���е�block����*block���̵߳�����
	//in[]�洢��ȫ���ڴ��� ����ָ�뱻ǡ���ض��룬����δ��뷢���ȫ���ڴ����񽫱��ϲ����⽫����޶ȵ�����ڴ����
	//Ҳ����˵һ��cuda�߳�Ҫȥ��η���ȫ���ڴ棬Ȼ�����Щֵ������
	//���ѭ��ʵ����Ҳ���������N��С������С��tid�������sum�г�ֵΪ0
	for (size_t i = blockIdx.x * blockDim.x + tid;
		i < N;
		i += blockDim.x * gridDim.x)
	{
		sum += in[i];
	}

	//ÿ���̰߳����õ����ۼ�ֵд�빲���ڴ�
	sPartials[tid] = sum;
	//��ִ�ж��������Ĺ�Լǰ����ͬ������
	__syncthreads();

	//blockSize������2�������η���ԭ�������ÿһ�ֶ�ֻ����һ��һ����̻߳��ڹ���
	//���ڹ����ڴ��е�ֵ ִ�ж��������Ĺ�Լ����
	//�����ڴ��к�벿�ֵ�ֵ����ӵ�ǰ�벿�ֵ�ֵ�ϣ�
	//����blockDim.x == 1024�����һ��activeThreads=512
	for (int activeThreads = blockDim.x >> 1;
		activeThreads;
		activeThreads >>= 1) //>>�Ƕ�������������� �ȼ�������2
							  //>>=�������Ҹ�ֵ����� Ҳ����activeThreads = activeThreads>>1
	{
		if (tid < activeThreads)
		{
			sPartials[tid] += sPartials[tid + activeThreads];
		}
		//ÿһ�ּ���֮��Ҫ�߳�ͬ��
		__syncthreads();
	}

	//ÿ��block��0���̴߳洢һ�������һ����numBlocks���̣߳����Դ洢����ô��������
	if (tid == 0)
	{
		out[blockIdx.x] = sPartials[0];
	}
}

//�����������kernel�����Ǳ����
//�ǳ���Ҫ ע������kernel�����Ĳ��� ��block��threads������==Reduction1_kernel�ڶ������������һ�����飩�ĳ��ȣ�Ҳ���ǹ����ڴ�sharedSize
void
Reduction_mean(int* answer,		//<out> ָ�����ս����ָ��
	int* partial,	//ָ��洢��ʱ���� �м������ָ�룬Ӧ���Ѿ����ٺ��˿ռ䡣����ĳ���Ӧ����blockDim.x
	const int* in, //�洢�������ݵ�ָ��
	size_t N,	//�������ݵ����� ������imgSizeRL
	int numBlocks, 
	int numThreads,
	hipStream_t& stream)
{
	unsigned int sharedSize = numThreads * sizeof(int);
	//��һ�εĽ��partialֻ��һ���м�������δ��ȫ����
	Reduction1_kernel <<<
		numBlocks, 
		numThreads, 
		sharedSize,
		stream>>> (
			partial,	//���ȵ���numThreads���м���partial�ĳ��ȸ�numThreads�йء�
			in,			//����ΪN
			N);

	//�ڶ��ν��answer�������յļ�������
	Reduction1_kernel <<<
		1, 
		numThreads, 
		sharedSize,
		stream>>> (
			answer,		//����Ϊ1
			partial,	//����ΪnumBlocks
			numBlocks);
}